#include "sm80_btn_gemm.cuh"
#include "conv_utils.cuh"
#include <random>
#include <iostream>

void benchmark_btngemm(
    int M, int N, int K,
    int run_time = 100
) {
    CUDA_CALL_CHECK(hipSetDevice(0));
    CUDA_CALL_CHECK(hipDeviceReset()); /* reset is needed to count overhead */

    int size_a = M*K/2;
    int size_b = K*N;
    int size_c = M*N;

    /* prepare host data */
    std::vector<int32_t> host_a(size_a, 0);    // input matrix a
    std::vector<int32_t> host_b(size_b, 0);    // input matrix b
    std::vector<int32_t> host_c_base(size_c, 0);      // output c
    std::vector<int32_t> host_c(size_c, 0);  

    /* generate host data */
    generate_random_array<int32_t>(host_a.data(), size_a, FP32);
    generate_random_array<int32_t>(host_b.data(), size_b, FP32);

    /* prepare device mem */
    int32_t* dev_a = 0;     // input matrix a
    int32_t* dev_b = 0;     // input matrix b
    int32_t* dev_c_base = 0;       // output c
    int32_t* dev_c = 0;       // output c

    /* allocate device memory */
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_a), size_a*sizeof(int32_t)));
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_b), size_b*sizeof(int32_t)));
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_c), size_c*sizeof(int32_t)));
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_c_base), size_c*sizeof(int32_t)));

    /* copy to device */
    CUDA_CALL_CHECK(hipMemcpy((void*)dev_a, (void*)host_a.data(), size_a*sizeof(int32_t), hipMemcpyHostToDevice));
    CUDA_CALL_CHECK(hipMemcpy((void*)dev_b, (void*)host_b.data(), size_b*sizeof(int32_t), hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // measure baseline performance
    float baseline_milliseconds = 0;
    hipEventRecord(start);
    for (int i = 0; i < run_time; i++) {
        sm80_btn::btn_gemm_baseline(
            dev_a, dev_b, dev_c_base, M, N, K/2
        );
    }
    hipEventRecord(stop);
    CUDA_CALL_CHECK(hipDeviceSynchronize());
    hipEventSynchronize(stop);
    hipEventElapsedTime(&baseline_milliseconds, start, stop);
    CUDA_CALL_CHECK(hipMemcpy((void*)host_c_base.data(), (void*)dev_c_base, size_c*sizeof(int32_t), hipMemcpyDeviceToHost));

    // measure optimized performance
    float optimize_milliseconds = 0;
    hipEventRecord(start);
    for (int i = 0; i < run_time; i++)
        sm80_btn::sm80_btn_gemm_multi_stage(
            dev_a, dev_b, dev_c, M, N, K/2
        );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    CUDA_CALL_CHECK(hipDeviceSynchronize());
    hipEventElapsedTime(&optimize_milliseconds, start, stop);
    CUDA_CALL_CHECK(hipMemcpy((void*)host_c.data(), (void*)dev_c, size_c*sizeof(int32_t), hipMemcpyDeviceToHost));

    // calculate throughput
    std::cout << "[" << M << "," << N << "," << K << "] ";
    std::cout << "Baseline:" << (float)baseline_milliseconds/run_time << "ms \t Optimize:" << (float)optimize_milliseconds/run_time << "ms\n";

    // host_cpu_c[1] = -1;
    int check = memcmp(host_c.data(), host_c_base.data(), size_c*sizeof(int32_t));
    int diff = 0;
    for (int i = 0; i < size_c; i++) {
        // std::cout << i << " " << host_c_base[i] << " " << host_c[i] << "\n";
        if (host_c_base[i] != host_c[i]) {
            // std::cout << "diff: " << i << " " << host_c_base[i] << " " << host_c[i] << "\n";
            diff++;
        }
    }
    std::cout << "Check=" << (check == 0) << " diff=" << diff << "\n";

}

int main(int argc, char *argv[]) {
    std::vector<int> mn = {16, 32, 64, 128, 256, 512, 1024, 2048, 4096};
    for (auto _mn : mn) {
        benchmark_btngemm(_mn, _mn, _mn*8, 10);
    }
}